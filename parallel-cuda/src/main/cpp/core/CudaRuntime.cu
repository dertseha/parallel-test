#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#include "cuda/CudaRuntime.h"

using cuda::CudaRuntime;
using gol::Buffer2d;

__device__ bool isAliveAt(ptrdiff_t x, ptrdiff_t y, uint8_t *image, size_t stride, size_t width, size_t height)
{
   size_t wrappedY = (y + height) % height;
   size_t wrappedX = (x + width) % width;
   uint8_t *row = image + (stride * wrappedY);
   return row[wrappedX] != 0x00;
}

__device__ size_t aliveCountAt(ptrdiff_t x, ptrdiff_t y, uint8_t *image, size_t stride, size_t width, size_t height)
{
   return isAliveAt(x, y, image, stride, width, height) ? 1 : 0;
}

__global__ void golKernel(uint8_t *input, size_t inputStride, uint8_t *output, size_t outputStride, size_t width, size_t height)
{
   ptrdiff_t x = blockIdx.x * blockDim.x + threadIdx.x;
   ptrdiff_t y = blockIdx.y * blockDim.y + threadIdx.y;

   bool selfAlive = isAliveAt(x, y, input, inputStride, width, height);
   size_t aliveNeighbors = //
      aliveCountAt(x - 1, y - 1, input, inputStride, width, height) + aliveCountAt(x, y - 1, input, inputStride, width, height) + aliveCountAt(x + 1, y - 1, input, inputStride, width, height) + //
      aliveCountAt(x - 1, y, input, inputStride, width, height) + aliveCountAt(x + 1, y, input, inputStride, width, height) + //
      aliveCountAt(x - 1, y + 1, input, inputStride, width, height) + aliveCountAt(x, y + 1, input, inputStride, width, height) + aliveCountAt(x + 1, y + 1, input, inputStride, width, height);
   bool shallBeAlive = (selfAlive && (aliveNeighbors == 2 || aliveNeighbors == 3)) || (!selfAlive && (aliveNeighbors == 3));

   uint8_t *row = output + outputStride * y;
   row[x] = shallBeAlive ? 0xFF : 0x00;
}

CudaRuntime::CudaRuntime()
{
}

CudaRuntime::~CudaRuntime()
{
   hipFree(input);
   hipFree(output);
}

void CudaRuntime::setInput(Buffer2d const &data)
{
   hipMallocPitch(&input, &inputStride, data.getWidth() * sizeof(uint8_t), data.getHeight());
   hipMemcpy2D(input, inputStride, data.getRow(0), data.getStride(), data.getWidth(), data.getHeight(), hipMemcpyHostToDevice);
   hipMallocPitch(&output, &outputStride, data.getWidth() * sizeof(uint8_t), data.getHeight());
   hipDeviceSynchronize();
   width = data.getWidth();
   height = data.getHeight();
}

void CudaRuntime::run()
{
   dim3 localSize(1, 1);
   dim3 globalSize((static_cast<decltype(localSize.x)>(width) + localSize.x - 1) / localSize.x,
                   (static_cast<decltype(localSize.y)>(height) + localSize.y - 1) / localSize.y);
   golKernel<<<globalSize, localSize>>>(input, inputStride, output, outputStride, width, height);
   hipDeviceSynchronize();
}

void CudaRuntime::getOutput(Buffer2d &data) const
{
   hipMemcpy2D(data.getRow(0), data.getStride(), output, outputStride, data.getWidth(), data.getHeight(), hipMemcpyDeviceToHost);
   hipDeviceSynchronize();
}
